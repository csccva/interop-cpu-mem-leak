#include "hip/hip_runtime.h"
//module load LUMI/24.03
//module load partition/G
//module load rocm
// hipcc -munsafe-fp-atomics --offload-arch=gfx90a -O3 main.cu -o hip.out
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
// #include <mpi.h>

#include <cstdio>
 #include <cstdlib>
#include <ctime>

#include <cmath>

#include <hip/hip_runtime.h>
#include <hipblas/hipblas.h>
#include <assert.h>
#include <hip/hip_complex.h>


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


#define NX 256
#define NY 256

void random_fill(double *array, int nx, int ny) {
    for (int i = 0; i < nx * ny; i++) {
        array[i] = (double)rand() / RAND_MAX;
    }
}

int main(int argc, char *argv[]) {
    double *A_h, *B_h, *C_h, *D_h;
    double *A_d, *B_d;
    int nx = NX, ny = NY;
    int rank, ntasks;
    size_t size = nx * ny * sizeof(double);

    // MPI Initialization
    // MPI_Init(&argc, &argv);
    // MPI_Comm_size(MPI_COMM_WORLD, &ntasks);
    // MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    // Allocate host memory
    A_h = (double *)malloc(size);
    B_h = (double *)malloc(size);
    C_h = (double *)malloc(size);
    D_h = (double *)malloc(size);

    // Fill A_h and B_h with random numbers
    random_fill(A_h, nx, ny);
    random_fill(B_h, nx, ny);

    // Set GPU device based on rank
    hipSetDevice(rank);

    // Create GPU stream
    hipStream_t gpu_stream;
    hipStreamCreate(&gpu_stream);

    for (int n_ii = 1; n_ii <= 10000000; n_ii++) {
        // Allocate device memory
        hipMallocAsync(&A_d, size, gpu_stream);
        hipMallocAsync(&B_d, size, gpu_stream);

        // Copy data from host to device
        hipMemcpyAsync(A_d, A_h, size, hipMemcpyHostToDevice, gpu_stream);
        hipMemcpyAsync(B_d, B_h, size, hipMemcpyHostToDevice, gpu_stream);

        // Copy data from device to host (for simplicity, copying back the same data)
        hipMemcpyAsync(D_h, B_d, size, hipMemcpyDeviceToHost, gpu_stream);
        hipMemcpyAsync(C_h, A_d, size, hipMemcpyDeviceToHost, gpu_stream);

        // Free device memory
        hipFreeAsync(A_d, gpu_stream);
        hipFreeAsync(B_d, gpu_stream);

        // Synchronize device
        gpuErrchk( hipDeviceSynchronize() ); //hipStreamSynchronize(gpu_stream);

        // Print the iteration number and difference sums
        double sum_A_diff = 0.0, sum_B_diff = 0.0;
        for (int i = 0; i < nx * ny; i++) {
            sum_A_diff += abs(A_h[i] - C_h[i]);
            sum_B_diff += abs(B_h[i] - D_h[i]);
        }
        printf("Iteration: %d, Sum_A_Diff: %f, Sum_B_Diff: %f\n", n_ii, sum_A_diff, sum_B_diff);
    }

    // Free host memory
    free(A_h);
    free(B_h);
    free(C_h);
    free(D_h);

    // Destroy GPU stream
    hipStreamDestroy(gpu_stream);

    // Finalize MPI
    // MPI_Finalize();

    return 0;
}
